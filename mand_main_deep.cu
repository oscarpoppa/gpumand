#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "mtypes.h"
#include "colors.h"
#include "bmp.h"
#include "aspect.h"
#include "get_coords.h"

#define BLOCK_SIZE 16

typedef struct {
    double2 real;
    double2 imag;
} Dcmp;

typedef struct {
    double2 llft;
    double ledg;
    size_t pitch;
    int colsz;
    int ilev;
} Init;

__global__ void MandKern(Dcmp* dev_cst_ptr, const uint32_t* dev_col_ptr, uint32_t* dev_pix_ptr, const Init* dev_init_ptr) {
    int cnt = 0; 
    const int iterations = dev_init_ptr->colsz * dev_init_ptr->ilev;
    const int pix_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int pix_y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ Dcmp row[BLOCK_SIZE];
    row[threadIdx.x].real.x = 0.0; 
    row[threadIdx.x].imag.x = 0.0; 
    row[threadIdx.x].real.y = 0.0;
    row[threadIdx.x].imag.y = 0.0; 
    Dcmp *locconst = (Dcmp*)((char*)dev_cst_ptr+pix_y*dev_init_ptr->pitch);
    locconst[pix_x].real.x = dev_init_ptr->llft.x;
    locconst[pix_x].real.y = dev_init_ptr->ledg*(double)pix_x/WIDTH; 
    locconst[pix_x].imag.x = dev_init_ptr->llft.y;
    locconst[pix_x].imag.y = dev_init_ptr->ledg*(double)pix_y/WIDTH;
    #pragma unroll 165
    for (; cnt<iterations; cnt++) {
        double nurb = pow(row[threadIdx.x].real.x, 2) - 
                      pow(row[threadIdx.x].imag.x, 2) + 
                      locconst[pix_x].real.x;
        double nurs = pow(row[threadIdx.x].real.y, 2) - 
                      pow(row[threadIdx.x].imag.y, 2) +
                      2.0 * row[threadIdx.x].real.y * row[threadIdx.x].real.x -
                      2.0 * row[threadIdx.x].imag.x * row[threadIdx.x].imag.y + 
                      locconst[pix_x].real.y;
        double nuib = 2.0 * row[threadIdx.x].real.x * row[threadIdx.x].imag.x + 
                      locconst[pix_x].imag.x;
        double nuis = 2.0 * row[threadIdx.x].real.x * row[threadIdx.x].imag.y + 
                      2.0 * row[threadIdx.x].real.y * row[threadIdx.x].imag.x + 
                      2.0 * row[threadIdx.x].real.y * row[threadIdx.x].imag.y + 
                      locconst[pix_x].imag.y;
        row[threadIdx.x].real.x = nurb;
        row[threadIdx.x].real.y = nurs;
        row[threadIdx.x].imag.x = nuib;
        row[threadIdx.x].imag.y = nuis;
        if (sqrt(pow(row[threadIdx.x].real.x+row[threadIdx.x].real.y, 2) + pow(row[threadIdx.x].imag.x+row[threadIdx.x].imag.y, 2)) > 2.0) 
            break;
    }
    if (cnt == iterations)
        dev_pix_ptr[WIDTH*pix_y+pix_x] = 0;
    else
        dev_pix_ptr[WIDTH*pix_y+pix_x] = dev_col_ptr[cnt/dev_init_ptr->ilev];
}

int main(int argc, char **argv) {
    Init istruct, *dev_init_ptr;
    size_t pitch;
    uint32_t *dev_pix_ptr, *dev_col_ptr;
    Dcmp* dev_cst_ptr;
    RunStart *init = get_coords(argc, argv);
    ColorInfo *colors = make_pall();
    istruct.colsz = colors->size;
    printf("%d\n" % istruct.colsz);
    istruct.llft.x = init->lleft.real;
    istruct.llft.y = init->lleft.imag;
    istruct.ledg = init->lleft.length;
    istruct.ilev = init->interleave;
    hipSetDevice(0);
    hipMallocPitch(&dev_cst_ptr, &pitch, WIDTH*sizeof(Dcmp), HEIGHT);
    istruct.pitch = pitch;
    hipMalloc(&dev_init_ptr, sizeof(Init));
    hipMalloc(&dev_pix_ptr, WIDTH*HEIGHT*sizeof(uint32_t));
    hipMalloc(&dev_col_ptr, colors->size*sizeof(uint32_t));
    hipMemcpy(dev_init_ptr, &istruct, sizeof(Init), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ptr, colors->pall, colors->size*sizeof(uint32_t), hipMemcpyHostToDevice);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(WIDTH/BLOCK_SIZE, HEIGHT/BLOCK_SIZE);
    MandKern<<<dimGrid, dimBlock>>>(dev_cst_ptr, dev_col_ptr, dev_pix_ptr, dev_init_ptr);
    uint32_t *pixarr = (uint32_t*)malloc(HEIGHT*WIDTH*sizeof(uint32_t));
    hipMemcpy(pixarr, dev_pix_ptr, HEIGHT*WIDTH*sizeof(uint32_t), hipMemcpyDeviceToHost); 
    gen_bmp(init->filename, pixarr, WIDTH, HEIGHT);
    hipFree(dev_cst_ptr);
    hipFree(dev_init_ptr);
    hipFree(dev_pix_ptr);
    hipFree(dev_col_ptr);
    free(pixarr);
    free(colors);
    free(init);
    return 0;
}

