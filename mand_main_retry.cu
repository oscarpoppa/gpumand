#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "mtypes.h"
#include "colors.h"
#include "bmp.h"
#include "aspect.h"
#include "get_coords.h"

#define BLOCK_SIZE 16

typedef double2 cudaDoubleComplex;

typedef struct {
    cudaDoubleComplex llft;
    double ledg;
    size_t pitch;
    int colsz;
    int ilev;
} Init;

__global__ void MandKern(cudaDoubleComplex* dev_cst_ptr, const uint32_t* dev_col_ptr, uint32_t* dev_pix_ptr, const Init* dev_init_ptr) {
    int cnt = 0; 
    const int iterations = dev_init_ptr->colsz * dev_init_ptr->ilev;
    const int pix_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int pix_y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ cudaDoubleComplex row[BLOCK_SIZE];
    row[threadIdx.x].x = 0.0; 
    row[threadIdx.x].y = 0.0;
    cudaDoubleComplex *locconst = (cudaDoubleComplex*)((char*)dev_cst_ptr+pix_y*dev_init_ptr->pitch);
    locconst[pix_x].x = dev_init_ptr->llft.x+dev_init_ptr->ledg*(double)pix_x/WIDTH; 
    locconst[pix_x].y = dev_init_ptr->llft.y+dev_init_ptr->ledg*(double)pix_y/WIDTH;
    #pragma unroll 165
    for (; cnt<iterations; cnt++) {
        double nux = row[threadIdx.x].x * row[threadIdx.x].x - row[threadIdx.x].y * row[threadIdx.x].y + locconst[pix_x].x;
        row[threadIdx.x].y = 2.0 * row[threadIdx.x].x * row[threadIdx.x].y + locconst[pix_x].y; 
        row[threadIdx.x].x = nux;
        if (sqrt(pow(row[threadIdx.x].y,2.0)+pow(row[threadIdx.x].y,2.0)) > 2.0)
            break; 
    }
    if (cnt == iterations)
        dev_pix_ptr[WIDTH*pix_y+pix_x] = 0;
    else
        dev_pix_ptr[WIDTH*pix_y+pix_x] = dev_col_ptr[cnt/dev_init_ptr->ilev];
}

int main(int argc, char **argv) {
    Init istruct, *dev_init_ptr;
    size_t pitch;
    uint32_t *dev_pix_ptr, *dev_col_ptr;
    cudaDoubleComplex* dev_cst_ptr;
    RunStart *init = get_coords(argc, argv);
    ColorInfo *colors = make_pall();
    istruct.colsz = colors->size;
    istruct.llft.x = init->lleft.real;
    istruct.llft.y = init->lleft.imag;
    istruct.ledg = init->lleft.length;
    istruct.ilev = init->interleave;
    hipSetDevice(0);
    hipMallocPitch(&dev_cst_ptr, &pitch, WIDTH*sizeof(cudaDoubleComplex), HEIGHT);
    istruct.pitch = pitch;
    hipMalloc(&dev_init_ptr, sizeof(Init));
    hipMalloc(&dev_pix_ptr, WIDTH*HEIGHT*sizeof(uint32_t));
    hipMalloc(&dev_col_ptr, colors->size*sizeof(uint32_t));
    hipMemcpy(dev_init_ptr, &istruct, sizeof(Init), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ptr, colors->pall, colors->size*sizeof(uint32_t), hipMemcpyHostToDevice);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(WIDTH/BLOCK_SIZE, HEIGHT/BLOCK_SIZE);
    MandKern<<<dimGrid, dimBlock>>>(dev_cst_ptr, dev_col_ptr, dev_pix_ptr, dev_init_ptr);
    uint32_t *pixarr = (uint32_t*)malloc(HEIGHT*WIDTH*sizeof(uint32_t));
    hipMemcpy(pixarr, dev_pix_ptr, HEIGHT*WIDTH*sizeof(uint32_t), hipMemcpyDeviceToHost); 
    gen_bmp(init->filename, pixarr, WIDTH, HEIGHT);
    hipFree(dev_cst_ptr);
    hipFree(dev_init_ptr);
    hipFree(dev_pix_ptr);
    hipFree(dev_col_ptr);
    free(pixarr);
    free(colors);
    free(init);
    return 0;
}

