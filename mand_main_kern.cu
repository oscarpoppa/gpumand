#include "hip/hip_runtime.h"
//$Id: mand_main_kern.cu,v 1.3 2017/12/06 20:51:29 dan Exp $
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <malloc.h>
#include "bmp.h"
#include "colors.h"
#include "mtypes.h"
#include "get_coords.h"
#include "aspect.h"
#define _block_size 20
#define cudaCheckError() { \
    hipError_t e=hipGetLastError(); \
    if(e!=hipSuccess) { \
        fprintf(stderr, "Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
        exit(0); \
    } \
}

__device__ 
void domult(Complex *ret, const Complex *cons) {
    double nureal, nuimag;
    nureal = pow(ret->real,2.0) - pow(ret->imag,2.0) + cons->real; 
    nuimag = 2.0 * ret->real * ret->imag + cons->imag;
    ret->imag = nuimag;
    ret->real = nureal;
    ret->length = sqrt(pow(nureal,2.0) + pow(nuimag,2.0));
}

__global__
void pixKern(uint32_t *pixarr, const Complex *log_lowleft, const uint32_t *colors, const uint32_t size, const uint32_t interleave) {
    uint32_t pix_x, pix_y, itr, iterations = size * interleave;
    double log_edge = log_lowleft->length;
    pix_x = blockIdx.x * blockDim.x + threadIdx.x;
    pix_y = blockIdx.y * blockDim.y + threadIdx.y;
    Complex ret = {0.0, 0.0, 0.0};
    Complex cons;
    cons.real = log_lowleft->real + (double)log_edge * (double)pix_x / _width;
    cons.imag = log_lowleft->imag + (double)log_edge * (double)pix_y / _width;
    cons.length = sqrt(pow(cons.real,2.0) + pow(cons.imag,2.0));
    for (itr=0; itr<iterations; itr++) {
        domult(&ret, &cons);
        if (ret.length > 2.0)
            break;
    }
    if (itr == iterations)
        pixarr[_width*pix_y+pix_x] = 0;
    else
        pixarr[_width*pix_y+pix_x] = colors[itr / interleave];
}

int main(int argc, char *argv[]) {
    const unsigned arlen = _width * _height * sizeof(uint32_t);
    uint32_t *d_colors, *d_pixarray, *pixarray = (uint32_t*)malloc(arlen);
    RunStart *init = get_coords(argc, argv);
    Complex *d_log_lowleft, *log_lowleft = (Complex*)malloc(sizeof(Complex));
    log_lowleft->real = init->lleft.real;
    log_lowleft->imag = init->lleft.imag;
    log_lowleft->length = init->lleft.length;
    ColorInfo *colors = make_pall(); 
    hipSetDevice(0);
    hipMalloc(&d_colors, colors->size * sizeof(uint32_t));
    hipMalloc(&d_log_lowleft, sizeof(Complex));
    hipMemcpy(d_log_lowleft, log_lowleft, sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(d_colors, colors->pall, colors->size * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMalloc(&d_pixarray, arlen);
    dim3 dimBlock(_block_size, _block_size);
    dim3 dimGrid(_width / _block_size, _height / _block_size);
    pixKern<<<dimGrid, dimBlock>>>(d_pixarray, d_log_lowleft, d_colors, colors->size, init->interleave);
    hipMemcpy(pixarray, d_pixarray, arlen, hipMemcpyDeviceToHost);
    hipFree(d_log_lowleft);
    hipFree(d_colors);
    free(log_lowleft);
    cudaCheckError();
    gen_bmp(init->filename, pixarray, _width, _height);
    hipFree(d_pixarray);
    free(pixarray);
    free(init);
    free(colors->pall);
    free(colors);
    return 0;
}

